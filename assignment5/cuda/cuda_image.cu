#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"

extern "C" {
#include "cuda_image.cuh"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ 
void grayscaleKernel(int pixels, int channels, byte *input, byte *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < pixels) {
        byte r = input[i * channels];
        byte g = input[i * channels + 1];
        byte b = input[i * channels + 2];

        // Use three to ignore the alpha channel
        byte gray = (r + g + b) / 3;
        output[i] = gray;
    }
}


/* Writes the results to image_t output */
extern "C"
void cuda_grayscale(image_t *input, image_t *output) {
    // How many bytes is the image
    int pixels = input->w * input->h;
    int bytes = pixels * input->n;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);

    output->data = (byte*)malloc(sizeof(byte) * pixels);
    // Create output image with the same dimensions
    output->w = input->w;
    output->h = input->h;
    // The gray output has only one channel
    output->n = 1;

    byte *d_in = NULL;
    byte *d_out = NULL;

    gpuErrchk(hipMalloc(&d_in, bytes * sizeof(byte))); 
    gpuErrchk(hipMalloc(&d_out, pixels * sizeof(byte)));
    gpuErrchk(hipMemcpy(d_in, input->data, bytes * sizeof(byte), hipMemcpyHostToDevice));

    grayscaleKernel<<<threadBlocks, threadBlockSize>>>(pixels, input->n, d_in, d_out);
    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(output->data, d_out, pixels * sizeof(byte), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));
}

__global__ 
void contrastKernel(int pixels, int mean, byte* data) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < pixels) { 
        if (data[i] > mean) {
            float d1 = (float)(data[i] - mean) / 255.0f;
            float d2 = 1.0f - ((float)mean / 255.0f);
            data[i] = (byte)((pow(d1, 0.5f) / pow(d2, 0.5f))* 255.0f);
        }
        else {
            data[i] = 0;
        }
    }
}

extern "C"
void cuda_contrast(image_t *image) {
    // Only use contrast on grayscale images
    if (image->n != 1) {
        return;
    }

    int brightness = 0;
    int pixels = image->w * image->h;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);
    
    for (int i = 0; i < pixels; i++) {
        brightness += image->data[i];
    }

    int mean = brightness / pixels;

    byte *device = NULL;

    gpuErrchk(hipMalloc(&device, pixels * sizeof(byte))); 
    gpuErrchk(hipMemcpy(device, image->data, pixels * sizeof(byte), hipMemcpyHostToDevice));

    contrastKernel<<<threadBlocks, threadBlockSize>>>(pixels, mean, device);
    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(image->data, device, pixels * sizeof(byte), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(device));
}

__device__
int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

__global__
void smoothingKernel(int pixels, int width, int height, byte *input, byte *output) {
    int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
    // The weights of the neighbourhood values, the sum is 81
    byte T[25] = {1, 2, 3, 2, 1, 2, 4, 6, 4, 2, 3, 6, 9, 6, 3, 2, 4, 6, 4, 2, 1, 2, 3, 2, 1};

    if (thread_index < pixels) {
        unsigned int sum = 0;
        // Loop over the neighbourhood
        for (int i = 0; i < 25; i++) {
                int row = i / 5;
                int column = i % 5;
                int index = mod(thread_index + column - 2  + (row - 2) * width, pixels);

                sum += T[row * 5 + column] * input[index];
        }
        output[thread_index] = sum / 81;
    }
}

extern "C"
void cuda_smoothing(image_t *image) {
    int pixels = image->w * image->h;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);
    
    byte *d_in = NULL;
    byte *d_out = NULL;
    
    gpuErrchk(hipMalloc(&d_in, pixels * sizeof(byte)));
    gpuErrchk(hipMalloc(&d_out, pixels * sizeof(byte))); 
    gpuErrchk(hipMemcpy(d_in, image->data, pixels * sizeof(byte), hipMemcpyHostToDevice));

    smoothingKernel<<<threadBlocks, threadBlockSize>>>(pixels, image->w, image->h, d_in, d_out);
    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(image->data, d_out, pixels * sizeof(byte), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));

}

extern "C"
void cuda_image(image_t *input, image_t *output) {
    cuda_grayscale(input, output);
    cuda_contrast(output);
    cuda_smoothing(output);
}