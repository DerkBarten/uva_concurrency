#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"
#include "openmp_image.h"

extern "C" {
#include "cuda_image.cuh"
}

__global__ 
void grayscaleKernel(int pixels, int channels, byte *input, byte *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < pixels) {
        byte r = input[i * channels];
        byte g = input[i * channels + 1];
        byte b = input[i * channels + 2];

        byte gray = (float)r * 0.299f + (float)g * 0.587f + (float)b * 0.114f;
        output[i] = gray;
    }
}


/* Writes the results to image_t output */
extern "C"
void cuda_grayscale(image_t *input, image_t *output, image_t *openmp_input, image_t *openmp_output) {
    // How many bytes is the image
    int pixels = input->w * input->h;
    int bytes = pixels * input->n;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);

    byte *d_in = NULL;
    byte *d_out = NULL;

    CUDA_CHECK(hipSetDevice, 0);

    CUDA_CHECK(hipMalloc, &d_in, bytes * sizeof(byte)); 
    CUDA_CHECK(hipMalloc, &d_out, pixels * sizeof(byte));
    CUDA_CHECK(hipMemcpyAsync, d_in, input->data, bytes * sizeof(byte), hipMemcpyHostToDevice);

    grayscaleKernel<<<threadBlocks, threadBlockSize, CUDA_DEFAULT_STREAM>>>(pixels, input->n, d_in, d_out);
    openmp_grayscale(openmp_input, openmp_output);
    //CUDA_CHECK(hipGetLastError);
    
    CUDA_CHECK(hipDeviceSynchronize);
    CUDA_CHECK(hipMemcpyAsync, output->data, d_out, pixels * sizeof(byte), hipMemcpyDeviceToHost);

    CUDA_CHECK(hipFree, d_in);
    CUDA_CHECK(hipFree, d_out);
}

__global__ 
void contrastKernel(int pixels, int mean, byte* data) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < pixels) { 
        if (data[i] > mean) {
            float d1 = (float)(data[i] - mean) / 255.0f;
            float d2 = 1.0f - ((float)mean / 255.0f);
            data[i] = (byte)((pow(d1, 0.5f) / pow(d2, 0.5f))* 255.0f);
        }
        else {
            data[i] = 0;
        }
    }
}

extern "C"
void cuda_contrast(image_t *image, image_t *openmp_input,int mean) {
    // Only use contrast on grayscale images
    if (image->n != 1) {
        return;
    }

    int pixels = image->w * image->h;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);
    
    byte *device = NULL;

    CUDA_CHECK(hipSetDevice, 0);

    CUDA_CHECK(hipMalloc, &device, pixels * sizeof(byte)); 
    CUDA_CHECK(hipMemcpyAsync, device, image->data, pixels * sizeof(byte), hipMemcpyHostToDevice);

    contrastKernel<<<threadBlocks, threadBlockSize, CUDA_DEFAULT_STREAM>>>(pixels, mean, device);
    
    //CUDA_CHECK(hipGetLastError);
    openmp_contrast(openmp_input, mean);
    
    CUDA_CHECK(hipDeviceSynchronize);
    CUDA_CHECK(hipMemcpyAsync, image->data, device, pixels * sizeof(byte), hipMemcpyDeviceToHost);

    CUDA_CHECK(hipFree, device);
}

__device__
int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

__global__
void smoothingKernel(int pixels, int width, int height, byte *input, byte *output) {
    int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
    // The weights of the neighbourhood values, the sum is 81
    byte T[25] = {1, 2, 3, 2, 1, 2, 4, 6, 4, 2, 3, 6, 9, 6, 3, 2, 4, 6, 4, 2, 1, 2, 3, 2, 1};

    int image_pixels = width * height;
    if (thread_index < pixels) {
        unsigned int sum = 0;
        // Loop over the neighbourhood
        for (int i = 0; i < 25; i++) {
                int row = i / 5;
                int column = i % 5;
                int index = mod(thread_index + column - 2  + (row - 2) * width, image_pixels);

                sum += T[row * 5 + column] * input[index];
        }
        output[thread_index] = sum / 81;
    }
}

// Need to add extra image for boundary conditions
extern "C"
void cuda_smoothing(image_t *image, image_t *openmp_input, image_t *original) {
    int pixels = image->w * image->h;
    // Add some extra read only pixels to the bottom pixels
    int padding = image->w * 3;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);
    
    byte *d_in = NULL;
    byte *d_out = NULL;
    
    CUDA_CHECK(hipSetDevice, 0);

    CUDA_CHECK(hipMalloc, &d_in, (pixels + padding) * sizeof(byte));
    CUDA_CHECK(hipMalloc, &d_out, pixels * sizeof(byte)); 
    CUDA_CHECK(hipMemcpyAsync, d_in, image->data, (pixels + padding) * sizeof(byte), hipMemcpyHostToDevice);

    smoothingKernel<<<threadBlocks, threadBlockSize, CUDA_DEFAULT_STREAM>>>(pixels, original->w, original->h, d_in, d_out);
    //CUDA_CHECK(hipGetLastError);
    openmp_smoothing(openmp_input, original);

    CUDA_CHECK(hipDeviceSynchronize);
    CUDA_CHECK(hipMemcpyAsync, image->data, d_out, pixels * sizeof(byte), hipMemcpyDeviceToHost);


    CUDA_CHECK(hipFree, d_in);
    CUDA_CHECK(hipFree, d_out);

   
}