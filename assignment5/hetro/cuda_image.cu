#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image.h"

extern "C" {
#include "cuda_image.cuh"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ 
void grayscaleKernel(int pixels, int channels, byte *input, byte *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < pixels) {
        byte r = input[i * channels];
        byte g = input[i * channels + 1];
        byte b = input[i * channels + 2];

        byte gray = (r + g + b) / 3;
        output[i] = gray;
    }
}


/* Writes the results to image_t output */
extern "C"
void rgb_to_grayscale(image_t *input, image_t *output) {
    // How many bytes is the image
    int pixels = input->w * input->h;
    int bytes = pixels * input->n;
    int threadBlockSize = 1024;
    int threadBlocks = ceil((float)pixels / (float)threadBlockSize);

    printf("Tread blocks: %i\n", threadBlocks);
    printf("Pixels: %i\n", pixels);
    printf("bytes: %i\n", bytes);
    printf("sizeof byte: %i\n", sizeof(byte));

    output->data = (byte*)malloc(sizeof(byte) * pixels);
    // Create output image with the same dimensions
    output->w = input->w;
    output->h = input->h;
    // The gray output has only one channel
    output->n = 1;

    byte *d_in = NULL;
    byte *d_out = NULL;

    gpuErrchk(hipMalloc(&d_in, bytes * sizeof(byte))); 
    gpuErrchk(hipMalloc(&d_out, pixels * sizeof(byte)));
    gpuErrchk(hipMemcpy(d_in, input->data, bytes * sizeof(byte), hipMemcpyHostToDevice));

    grayscaleKernel<<<threadBlocks, threadBlockSize>>>(pixels, input->n, d_in, d_out);

    // Wait for all kernels to finish
    gpuErrchk(hipDeviceSynchronize());

    // Assuming output->data has enough memory allocated
    gpuErrchk(hipMemcpy(output->data, d_out, pixels * sizeof(byte), hipMemcpyDeviceToHost));

    // for (int i = 0; i < 100; i++) {
    //     printf("%i\n", output->data[i]);
    // }

    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));
}
